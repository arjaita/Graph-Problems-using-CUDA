#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <malloc.h>
#include <assert.h>
#include <stdio.h>
#include "myHash.h"
#include "node.h"



using namespace std;

#define SIZE 362880
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__global__
void dum(){

}

__device__ bool *frontier;	//Fa
__device__ bool *Ufrontier;	//Fua
__device__ bool *V;	//Xa


//node *visited[SIZE]; //could not create
__device__ node **visited;

__device__ node *initial, *goal;
__device__ int gIndex;


//__device__ bool fin;

//__global__ void initBFS(bool *fin) {
__global__ void initBFS() {

	int index;
	index = initial->getIndex();
	frontier[index] = true; //enQueu
	visited[index] = initial; //mark as visited
	V[index] = true;
	//*fin = false;
}

__device__
void clear() {
	delete[] visited;

	//add......
}

__global__
void createBFS(){
	node *x,*adj;
//	printf("\nin bfs\n");
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	int k=5;
//	if(tid < SIZE && tid == gIndex){
//		*l = 64;
//	}
	int index;
//	do{
//		if(tid == 45507)
//			printf("^^^%d:%d",k,frontier[tid]);

		if(tid < SIZE && frontier[tid]){	//deQueue in parallel
//			printf("\nin bfs true\n");
			frontier[tid] = false;
			x = visited[tid];
//			if(k == 3)
//				printf("^^^%d",tid);
//			printf("%d:%s\n",tid,x->state.a);
//
//			if(tid == gIndex){	//Reached Goal
//				*fin = true;
//				*l = x->depth;
//				continue;
//			}else{
				for (int i = UP; i <= RIGHT; ++i) {
					adj = x->move((Move)i);
//					if(tid == 46227)
//						printf("\n%d:",i);
					if(adj == NULL)
						continue;
					index = adj->getIndex();
//					if(tid == 46227)
//						printf("%d",index);
					if(!V[index]){
						x->child[i] = adj;
						x->child[i]->depth = x->depth + 1;
						x->child[i]->parent = x;		//mark path
						visited[index] = x->child[i];	//mark as visited
						Ufrontier[index] = true;			//for each child enQueue and mark
//						if(tid == 46227)
//							printf("*****%d",frontier[index]);
					}else{
						x->child[i] = NULL;
					}

				}	//end for
		}	//end if
		//assert(!fin);
//		__syncthreads();
//		__threadfence_system();

		/*
		 * __synctreads() synchronizes all the threads in a block.
		 * there is no way to synchronize threads across blocks
		 * Inter-block GPU communication via fast barrier synchronization" from 2010 by Xiao and Feng (Virginia Tech) is a nice solution
		 * http://scholar.google.com/scholar?cluster=4900456939806066632
		 *
		 * The usual way to sync all threads in all blocks is to call two separate kernels
		 * The second kernel will not be run until the first has completed
		 *
		 * __threadfence_system()
		 */
//	}while(k--);

	//clear();
}	//end function


__global__
void k2(bool *g_over,int *l){


	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	printf("\nin k2\n");
	if(tid < SIZE && Ufrontier[tid]){
		frontier[tid] = true;
		V[tid] = true;
		//g_over = true;
//		printf("tid = %d , gindex = %d",tid,gIndex);
		if(tid == gIndex){	//Reached Goal
				*g_over = true;
				*l = visited[tid]->depth;
//				printf("\n in loop");
		}
		Ufrontier[tid] = false;
	}
}

__global__
void clearArray(){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < SIZE){
		 frontier[tid] = false;	//Fa
		 Ufrontier[tid] = false;	//Fua
		 visited[tid]=NULL;
		 V[tid] = false;	//Xa


	}
}

__global__
void initialize(){
	initial = new node;
	initial->state.a[0] = '0';
	initial->state.a[1] = '1';
	initial->state.a[2] = '3';
	initial->state.a[3] = '4';
	initial->state.a[4] = '5';
	initial->state.a[5] = '6';
	initial->state.a[6] = '7';
	initial->state.a[7] = '8';
	initial->state.a[8] = '2';
	initial->state.pos = 0;
	initial->depth = 0;


	goal = new node;
	goal->state.a[0] = '5';
	goal->state.a[1] = '4';
	goal->state.a[2] = '3';
	goal->state.a[3] = '7';
	goal->state.a[4] = '8';
	goal->state.a[5] = '6';
	goal->state.a[6] = '1';
	goal->state.a[7] = '0';
	goal->state.a[8] = '2';
	goal->state.pos = 7;
	gIndex = goal->getIndex();


	visited = new node*[SIZE];	//
	frontier = new bool[SIZE];	//Fa
	Ufrontier = new bool[SIZE];	//Fua
	V = new bool[SIZE];	//Xa

}



int main(int argc, char **argv) {
//	time_t t1,t2;
//	time(&t1);
	clock_t t;
	t = clock();

//	cout<<"\n1\n";

	cout<<(float(clock()-t))/CLOCKS_PER_SEC<<endl;
	hipDeviceProp_t dev;
	hipGetDeviceProperties(&dev,0);
	int maxT = dev.maxThreadsPerBlock;
	cout<<(float(clock()-t))/CLOCKS_PER_SEC<<endl;

int number_of_blocks = (SIZE + maxT -1)/maxT;
int threadsPerBlock = maxT;

dum<<<1,1>>>();
	int len,*d_len;
	//len = (int*)malloc(sizeof(int));
	hipMalloc((void**)&d_len,sizeof(int));
	cout<<(float(clock()-t))/CLOCKS_PER_SEC<<endl;

	initialize<<<1,1>>>();
//	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	cout<<(float(clock()-t))/CLOCKS_PER_SEC<<endl;

	clearArray<<<number_of_blocks, threadsPerBlock>>>();
//	CUDA_CHECK_RETURN(hipDeviceSynchronize());

//bool fin = false,*d_fin;
//hipMalloc((void**)&d_fin, sizeof(bool));

	initBFS<<<1,1>>>();

//	cout<<"\n2\n";



	cout<<(float(clock()-t))/CLOCKS_PER_SEC<<endl;
t = clock();
//cout<<"\n3\n";
//modification
int k=0;
bool stop = false;
bool *d_over;

hipMalloc((void**) &d_over,sizeof(bool));
hipMemcpy(d_over, &stop, sizeof(bool),hipMemcpyHostToDevice);
//cout<<"\n4\n";
do{
//	stop = false;
//	hipMemcpy(d_over, &stop, sizeof(bool),hipMemcpyHostToDevice);

	createBFS<<<number_of_blocks, threadsPerBlock>>>();	//k1
//	CUDA_CHECK_RETURN(hipDeviceSynchronize());

//	cout<<"\n5\n";
	k2<<<number_of_blocks, threadsPerBlock>>>(d_over,d_len);
//	cout<<"\n6\n";
	//hipMemcpy( &stop, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
	//cout<<d_over;
	CUDA_CHECK_RETURN(hipMemcpy(&stop,d_over,sizeof(bool),hipMemcpyDeviceToHost));
//	cout<<"stop = "<<stop<<"\n";


	//CUDA_CHECK_RETURN(hipMemcpy(&stop,d_over,sizeof(bool),hipMemcpyDeviceToHost));
//	cout<<"****"<<endl;

	k++;
//	if(k==15)
//	{
//		printf("\nbreak fourcefully\n");
//		break;
//	}
}while(!stop);
	//cout<<(float(clock()-t))/CLOCKS_PER_SEC<<endl;
//	CUDA_CHECK_RETURN(hipGetLastError());
	//CUDA_CHECK_RETURN(hipMemcpy(len,d_len,sizeof(int),hipMemcpyDeviceToHost));

	hipMemcpy( &len, d_len, sizeof(int), hipMemcpyDeviceToHost);
	cout<<"\nlen = "<<len<<endl;

//	time(&t2);
//
//	cout<<difftime(t2,t1);
	cout<<(float(clock()-t))/CLOCKS_PER_SEC<<endl;

	cout<<"Kernel executed "<<k<<"times";

//	hipFree(frontier);
//	hipFree(Ufrontier);
//	hipFree(V);

}



